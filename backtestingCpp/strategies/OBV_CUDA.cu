#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/count.h>
#include <thrust/scan.h>
#include <thrust/functional.h>

// Define a functor to calculate the mean
struct MeanFunctor
{
    const int window_size;

    MeanFunctor(int size) : window_size(size) {}

    __host__ __device__
    double operator()(const double& x) const
    {
        return x / window_size;
    }
};

void Obv::execute_backtest()
{
    pnl = 0.0;
    max_dd = 0.0;

    double max_pnl = 0.0;
    int current_position = 0;
    double entry_price;

    thrust::device_vector<double> obv_closes(close.begin(), close.end());

    // Create a temporary vector to store the mean values
    thrust::device_vector<double> mean_values(obv_closes.size());

    // Calculate the mean values using transform_reduce
    thrust::transform_reduce(obv_closes.begin(), obv_closes.end(), mean_values.begin(), MeanFunctor(10),0.0, thrust::plus<double>());

    // Create a temporary vector to store the cumulative sum
    thrust::device_vector<double> cumulative_sum(obv_closes.size());

    // Calculate the cumulative sum using inclusive_scan
    thrust::inclusive_scan(obv_closes.begin(), obv_closes.end(), cumulative_sum.begin());

    for (int i = 0; i < obv_closes.size(); i++)
    {
        if (obv_closes.size() > 10) {
            obv_closes.erase(obv_closes.begin());
        }

        if (obv_closes.size() < 10) {
            continue;
        }

        double mean = mean_values[i];

        // Long Signal
        if (close[i] > mean && current_position <= 0) {

            if (current_position == -1) {
                double pnl_temp = (entry_price / close[i] - 1) * 100;
                pnl += pnl_temp;
                max_pnl = max(max_pnl, pnl);
                max_dd = max(max_dd, max_pnl - pnl);
            }

            current_position = 1;
            entry_price = close[i];
        }

        // Short Signal
        if (close[i] < mean && current_position >= 0) {

            if (current_position == 1) {
                double pnl_temp = (close[i] / entry_price - 1) * 100;
                pnl += pnl_temp;
                max_pnl = max(max_pnl, pnl);
                max_dd = max(max_dd, max_pnl - pnl);
            }

            current_position = -1;
            entry_price = close[i];
        }
    }
}